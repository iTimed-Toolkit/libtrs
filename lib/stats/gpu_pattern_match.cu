#include "hip/hip_runtime.h"
#include "statistics.h"

#include "__trace_internal.h"

#include <stdlib.h>

#define BLOCKDIM    128

__global__ void __do_gpu_pattern_match(const float *data, int data_len,
                                       const float *pattern, int pattern_len,
                                       float s_pattern, float *res)
{
    int i;
    size_t block_x, my_x;

    // for various data values
    float data_val, m_data,
            m_new_data, s_data;

    // for accumulated covariance
    float cov, pearson, len, p;

    block_x = blockDim.x * blockIdx.x;
    my_x = block_x + threadIdx.x;

    // length of data might not be evenly divisible by BLOCKDIM
    if(my_x < (data_len - pattern_len))
    {
        // unrolled first iteration (different)
        m_data = data[my_x];
        s_data = 0.0f;
        cov = 0.0f;

        i = 1;
        while(i < pattern_len)
        {
            p = pattern[i];
            data_val = data[my_x + i];

            // update covariance and related stats
            m_new_data = m_data + (data_val - m_data) / (float) (i + 1);
            s_data += ((data_val - m_data) * (data_val - m_new_data));
            cov += ((data_val - m_data) * p);
            m_data = m_new_data;
            i++;
        }

        len = (float) (pattern_len - 1);
        pearson = cov / (len * sqrtf(s_data / len) * sqrtf(s_pattern / len));
        res[my_x] = pearson;
    }
}

int gpu_pattern_preprocess(float *pattern, int pattern_len, float **out, float *var)
{
    int i;
    hipError_t ret;
    float val, s,
            m, m_new, *res, *res_gpu;

    res = (float *) calloc(pattern_len, sizeof(float));
    if(!res)
    {
        err("Failed to allocate result array\n");
        return -ENOMEM;
    }

    ret = hipMalloc(&res_gpu, pattern_len * sizeof(float));
    if(ret != hipSuccess)
    {
        err("Failed to allocate result array on GPU: %s\n", hipGetErrorName(ret));
        free(res);
        return -EINVAL;
    }

    m = pattern[0];
    s = 0;
    res[0] = 0.0f;

    for(i = 1; i < pattern_len; i++)
    {
        val = pattern[i];

        m_new = m + (val - m) / (float) (i + 1);
        s += ((val - m) * (val - m_new));
        m = m_new;

        res[i] = (val - m_new);
    }

    ret = hipMemcpy(res_gpu, res, pattern_len * sizeof(float), hipMemcpyHostToDevice);
    if(ret != hipSuccess)
    {
        err("Failed to move preprocessed pattern to GPU\n");
        hipFree(res_gpu);
        free(res);
        return -EINVAL;
    }

    free(res);
    *out = res_gpu;
    *var = s;
    return 0;

}

int gpu_pattern_free(float *pattern)
{
    hipFree(pattern);
    return 0;
}

int gpu_pattern_match(float *data, int data_len, float *pattern, int pattern_len, float s_pattern, float **pearson)
{
    int ret;
    float *data_gpu = NULL, *res_gpu = NULL, *res = NULL;

    hipError_t cuda_ret;
    dim3 grid, block;

    hipStream_t stream;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    res = (float *) calloc(data_len - pattern_len, sizeof(float));
    if(!res)
    {
        err("Failed to allocate result array\n");
        return -ENOMEM;
    }

    cuda_ret = hipStreamCreate(&stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to create cuda stream: %s\n", hipGetErrorName(cuda_ret));
        free(res);
        ret = -EINVAL;
        goto __done;
    }

    cuda_ret = hipMallocAsync(&data_gpu, data_len * sizeof(float), stream);
    if(cuda_ret == hipSuccess)
        cuda_ret = hipMallocAsync(&res_gpu, (data_len - pattern_len) * sizeof(float), stream);
    if(cuda_ret == hipSuccess)
        cuda_ret = hipMemcpyAsync(data_gpu, data, data_len * sizeof(float), hipMemcpyHostToDevice, stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to initialize some GPU data: %s\n", hipGetErrorName(cuda_ret));
        ret = -EINVAL;
        goto __free_gpu;
    }

    block.x = BLOCKDIM;
    grid.x = (data_len / BLOCKDIM) + 1;

    hipEventRecord(start, stream);

    __do_gpu_pattern_match<<<grid, block, 0, stream>>>(data_gpu, data_len, pattern, pattern_len, s_pattern, res_gpu);
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    debug("gpu elapsed %f ms\n", time);
    cuda_ret = hipMemcpyAsync(res, res_gpu, (data_len - pattern_len) * sizeof(float), hipMemcpyDeviceToHost, stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to retrieve results from GPU: %s\n", hipGetErrorName(cuda_ret));
        ret = -EINVAL;
        goto __free_gpu;
    }

    *pearson = res;
    ret = 0;
__free_gpu:
    if(data_gpu)
        hipFreeAsync(data_gpu, stream);

    if(res_gpu)
        hipFreeAsync(res_gpu, stream);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
__done:
    return ret;
}
