#include "hip/hip_runtime.h"
#include "statistics.h"
#include "__stat_internal.h"
#include "__trace_internal.h"

#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#define THREADS_PER_BLOCK       32

struct dual_array_gpu_vars
{
    bool host_stale;
    hipStream_t stream;
    float *val0, *val1, *m, *s, *cov;
};

__global__ void __do_accumulate_dual_array_val1(float count,
                                                float *val1, int len0, int len1,
                                                float *m, float *s)
{
    unsigned int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    float v1, m1, m1_new, ds1;

    if(id_x < len1)
    {
        v1 = val1[id_x];
        m1 = m[len0 + id_x];
        m1_new = m1 + (v1 - m1) / count;
        ds1 = ((v1 - m1) * (v1 - m1_new));
        m[len0 + id_x] = m1_new;
        s[len0 + id_x] += ds1;
    }
}

__global__ void __do_accumulate_dual_array_cov(float count,
                                               float *val0, float *val1,
                                               int len0, int len1,
                                               float *m, float *s, float *cov)
{
    unsigned int id0 = blockDim.x * blockIdx.x + threadIdx.x,
            id1 = blockDim.y * blockIdx.y + threadIdx.y;
    float v0, m0, v1, m1_new, dcov;

    if(id0 < len0 && id1 < len1)
    {
        v0 = val0[id0];
        m0 = m[id0];
        v1 = val1[id1];
        m1_new = m[len0 + id1];

        dcov = ((v0 - m0) * (v1 - m1_new));
        cov[len0 * id1 + id0] += dcov;
    }
}

__global__ void __do_accumulate_dual_array_val0(float count,
                                                float *val0, int len0,
                                                float *m, float *s)
{
    unsigned int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    float v0, m0, m0_new, ds0;

    if(id_x < len0)
    {
        v0 = val0[id_x];
        m0 = m[id_x];
        m0_new = m0 + (v0 - m0) / count;
        ds0 = ((v0 - m0) * (v0 - m0_new));
        m[id_x] = m0_new;
        s[id_x] += ds0;
    }
}

__global__ void __do_init_dual_array(float *val0, float *val1,
                                     int len0, int len1,
                                     float *m, float *s, float *cov)
{
    unsigned int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    if(id_x < len0)
    {
        m[id_x] = val0[id_x];
        s[id_x] = 0;
        cov[id_x] = 0;
    }
    else if(id_x < len0 + len1)
    {
        m[id_x] = val1[id_x - len0];
        s[id_x] = 0;
        cov[id_x] = 0;
    }
}

int gpu_sync_dual_array(struct accumulator *acc)
{
    hipError_t cuda_ret;
    struct dual_array_gpu_vars *vars =
            (struct dual_array_gpu_vars *) acc->gpu_vars;

    if(vars->host_stale)
    {
        cuda_ret = hipMemcpyAsync(acc->_AVG.a, vars->m,
                                   (acc->dim0 + acc->dim1) * sizeof(float),
                                   hipMemcpyDeviceToHost,
                                   vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to copy m from GPU to host: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        cuda_ret = hipMemcpyAsync(acc->_DEV.a, vars->s,
                                   (acc->dim0 + acc->dim1) * sizeof(float),
                                   hipMemcpyDeviceToHost,
                                   vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to copy s from GPU to host: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        cuda_ret = hipMemcpyAsync(acc->_COV.a, vars->cov,
                                   (acc->dim0 * acc->dim1) * sizeof(float),
                                   hipMemcpyDeviceToHost,
                                   vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to copy cov from GPU to host: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        cuda_ret = hipStreamSynchronize(vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to synchronize stream: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        vars->host_stale = false;
    }

    return 0;
}

int gpu_accumulate_dual_array(struct accumulator *acc, float *val0, float *val1, int len0, int len1)
{
    hipError_t cuda_ret;
    struct dual_array_gpu_vars *vars =
            (struct dual_array_gpu_vars *) acc->gpu_vars;

    dim3 block(THREADS_PER_BLOCK);
    dim3 block2(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    cuda_ret = hipMemcpyAsync(vars->val0, val0,
                               len0 * sizeof(float),
                               hipMemcpyHostToDevice,
                               vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to memcpy val0 to GPU: %s\n", hipGetErrorName(cuda_ret));
        return -EINVAL;
    }

    cuda_ret = hipMemcpyAsync(vars->val1, val1,
                               len1 * sizeof(float),
                               hipMemcpyHostToDevice,
                               vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to memcpy val1 to GPU: %s\n", hipGetErrorName(cuda_ret));
        return -EINVAL;
    }

    acc->count++;
    if(acc->count == 1)
    {
        dim3 grid((len0 + len1) / THREADS_PER_BLOCK + 1);
        __do_init_dual_array<<<grid, block, 0, vars->stream>>>(vars->val0, vars->val1,
                                                               len0, len1,
                                                               vars->m, vars->s, vars->cov);
    }
    else
    {
        dim3 grid_val1(len1 / THREADS_PER_BLOCK + 1);
        __do_accumulate_dual_array_val1<<<grid_val1, block, 0, vars->stream>>>(acc->count, vars->val1, len0, len1,
                                                                               vars->m, vars->s);
        dim3 grid_cov(len0 / THREADS_PER_BLOCK + 1, len1 / THREADS_PER_BLOCK + 1);
        __do_accumulate_dual_array_cov<<<grid_cov, block2, 0, vars->stream>>>(acc->count, vars->val0, vars->val1,
                                                                              len0, len1,
                                                                              vars->m, vars->s, vars->cov);
        dim3 grid_val0(len0 / THREADS_PER_BLOCK + 1);
        __do_accumulate_dual_array_val0<<<grid_val0, block, 0, vars->stream>>>(acc->count, vars->val0, len0,
                                                                               vars->m, vars->s);
    }

    vars->host_stale = true;
    return 0;
}

int gpu_init_dual_array(struct accumulator *acc, int num0, int num1)
{
    int ret;
    hipError_t cuda_ret;
    struct dual_array_gpu_vars *vars;

    if(!acc)
    {
        err("Invalid accumulator\n");
        return -EINVAL;
    }

    vars = (struct dual_array_gpu_vars *)
            calloc(1, sizeof(struct dual_array_gpu_vars));
    if(!vars)
    {
        err("Failed to allocate memory for GPU vars\n");
        return -ENOMEM;
    }

    vars->host_stale = false;

    cuda_ret = hipStreamCreate(&vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to create cuda stream: %s\n", hipGetErrorName(cuda_ret));
        ret = -EINVAL;
        goto __free_vars;
    }

    cuda_ret = hipMallocAsync(&vars->val0, num0 * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU val0 array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __destroy_stream;
    }

    cuda_ret = hipMallocAsync(&vars->val1, num1 * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU val1 array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    cuda_ret = hipMallocAsync(&vars->m, (num0 + num1) * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU m array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    cuda_ret = hipMallocAsync(&vars->s, (num0 + num1) * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU s array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    cuda_ret = hipMallocAsync(&vars->cov, (num0 * num1) * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU cov array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    acc->gpu_vars = (void *) vars;
    return 0;

__free_bufs:
    if(vars->val0)
        hipFreeAsync(vars->val0, vars->stream);

    if(vars->val1)
        hipFreeAsync(vars->val1, vars->stream);

    if(vars->m)
        hipFreeAsync(vars->m, vars->stream);

    if(vars->s)
        hipFreeAsync(vars->s, vars->stream);

    if(vars->cov)
        hipFreeAsync(vars->cov, vars->stream);

__destroy_stream:
    hipStreamDestroy(vars->stream);

__free_vars:
    free(vars);
    return ret;
}

int gpu_free_dual_array(struct accumulator *acc)
{
    struct dual_array_gpu_vars *vars =
            (struct dual_array_gpu_vars *) acc->gpu_vars;

    if(vars->val0)
        hipFreeAsync(vars->val0, vars->stream);
    if(vars->val1)
        hipFreeAsync(vars->val1, vars->stream);
    if(vars->m)
        hipFreeAsync(vars->m, vars->stream);
    if(vars->s)
        hipFreeAsync(vars->s, vars->stream);
    if(vars->cov)
        hipFreeAsync(vars->cov, vars->stream);

    hipStreamDestroy(vars->stream);
    free(vars);
    return 0;
}
