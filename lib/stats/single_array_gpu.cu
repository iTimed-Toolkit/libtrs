#include "hip/hip_runtime.h"
#include "__stat_internal.h"

__global__ void __do_accumulate_single_array(float count, float *val, int len, float *m, float *s)
{
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    if(id_x < len)
    {

    }
}

__global__ void __do_init_single_array(float *val, int len, float *m, float *s)
{
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    if(id_x < len)
    {
        m[id_x] = val[id_x];
        s[id_x] = 0;
    }
}


int __accumulate_single_array_gpu(struct accumulator *acc, float *val, int len)
{
    acc->count++;
    if(acc->count == 1)
    {

    }
}