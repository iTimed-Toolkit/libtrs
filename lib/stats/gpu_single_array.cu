#include "hip/hip_runtime.h"
#include "__stat_internal.h"
#include "__trace_internal.h"

#include <errno.h>
#define THREADS_PER_BLOCK       32

struct single_array_gpu_vars
{
    bool host_stale;
    hipStream_t stream;
    float *val, *m, *s;
};

__global__ void __do_accumulate_single_array(float count, float *val, int len, float *m, float *s)
{
    unsigned int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    float my_val, my_m, my_s, my_new_m, my_new_s;

    if(id_x < len)
    {
        my_val = val[id_x];
        my_m = m[id_x];
        my_s = s[id_x];

        my_new_m = my_m + (my_val - my_m) / count;
        my_new_s = my_s + ((my_val - my_m) * (my_val - my_new_m));

        m[id_x] = my_new_m;
        s[id_x] = my_new_s;
    }
}

__global__ void __do_init_single_array(float *val, int len, float *m, float *s)
{
    unsigned int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    if(id_x < len)
    {
        m[id_x] = val[id_x];
        s[id_x] = 0;
    }
}

int __sync_single_array_gpu(struct accumulator *acc)
{
    hipError_t cuda_ret;
    struct single_array_gpu_vars *vars =
            (struct single_array_gpu_vars *) acc->gpu_vars;

    if(vars->host_stale)
    {
        cuda_ret = hipMemcpyAsync(acc->m.a, vars->m,
                                   acc->dim0 * sizeof(float),
                                   hipMemcpyDeviceToHost,
                                   vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to copy m from GPU to host: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        cuda_ret = hipMemcpyAsync(acc->s.a, vars->s,
                                   acc->dim0 * sizeof(float),
                                   hipMemcpyDeviceToHost,
                                   vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to copy s from GPU to host: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        cuda_ret = hipStreamSynchronize(vars->stream);
        if(cuda_ret != hipSuccess)
        {
            err("Failed to synchronize stream: %s\n", hipGetErrorName(cuda_ret));
            return -EINVAL;
        }

        vars->host_stale = false;
    }

    return 0;
}

int __accumulate_single_array_gpu(struct accumulator *acc, float *val, int len)
{
    hipError_t cuda_ret;
    struct single_array_gpu_vars *vars =
            (struct single_array_gpu_vars *) acc->gpu_vars;

    dim3 grid(len / THREADS_PER_BLOCK + 1);
    dim3 block(THREADS_PER_BLOCK);

    acc->count++;
    cuda_ret = hipMemcpyAsync(vars->val, val,
                               len * sizeof(float),
                               hipMemcpyHostToDevice,
                               vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to memcpy values to GPU: %s\n", hipGetErrorName(cuda_ret));
        return -EINVAL;
    }

    if(acc->count == 1)
        __do_init_single_array<<<grid, block, 0, vars->stream>>>(vars->val, len, vars->m, vars->s);
    else
        __do_accumulate_single_array<<<grid, block, 0, vars->stream>>>(acc->count, vars->val,
                                                                       len, vars->m, vars->s);

    vars->host_stale = true;
    return 0;
}

int __init_single_array_gpu(struct accumulator *acc, int num)
{
    int ret;
    hipError_t cuda_ret;
    struct single_array_gpu_vars *vars;

    if(!acc)
    {
        err("Invalid accumulator\n");
        return -EINVAL;
    }

    vars = (struct single_array_gpu_vars *)
            calloc(1, sizeof(struct single_array_gpu_vars));
    if(!vars)
    {
        err("Failed to allocate memory for GPU vars\n");
        return -ENOMEM;
    }

    vars->host_stale = false;

    cuda_ret = hipStreamCreate(&vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to create cuda stream: %s\n", hipGetErrorName(cuda_ret));
        ret = -EINVAL;
        goto __free_vars;
    }

    cuda_ret = hipMallocAsync(&vars->val, num * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU val array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __destroy_stream;
    }

    cuda_ret = hipMallocAsync(&vars->m, num * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU m array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    cuda_ret = hipMallocAsync(&vars->s, num * sizeof(float), vars->stream);
    if(cuda_ret != hipSuccess)
    {
        err("Failed to allocate GPU s array: %s\n", hipGetErrorName(cuda_ret));
        ret = -ENOMEM;
        goto __free_bufs;
    }

    acc->gpu_vars = (void *) vars;
    return 0;

__free_bufs:
    if(vars->val)
        hipFreeAsync(vars->val, vars->stream);

    if(vars->m)
        hipFreeAsync(vars->m, vars->stream);

    if(vars->s)
        hipFreeAsync(vars->s, vars->stream);

__destroy_stream:
    hipStreamDestroy(vars->stream);

__free_vars:
    free(vars);
    return ret;
}
